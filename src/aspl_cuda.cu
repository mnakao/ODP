#include "hip/hip_runtime.h"
#include "common.h"
static uint64_t *_A_dev, *_B_dev;
static uint64_t *_result, *_result_dev;
static int *_adjacency_dev, *_num_degrees_dev = NULL, *_itable = NULL, *_itable_dev = NULL;
static bool _is_profile = false, _enable_grid_s = false;
static int _nodes, _degree, _symmetries, _kind, _height = -1;
static double _mem_usage, _elapsed_time;
static unsigned int _times;

extern "C" bool ODP_Check_profile();
extern "C" double ODP_Get_time();
extern "C" void ODP_Create_itable(const int width, const int height, const int symmetries, int *itable);
extern "C" int ODP_LOCAL_INDEX_GRID(const int x, const int width, const int height, const int symmetries);
extern "C" int ODP_ROTATE(const int v, const int width, const int height, const int symmetries, const int degree);
extern "C" void ODP_Profile(const char* name, const int kind, const int symmetries, const double mem_usage,
			    const double elapsed_time, const unsigned int times, const int procs);
extern "C" int ODP_Get_kind(const int nodes, const int degree, const int* num_degrees, const int symmetries,
			    const int procs, const bool is_cpu, const bool enable_grid_s);
extern "C" double ODP_Get_mem_usage(const int kind, const int nodes, const int degree, const int symmetries,
				    const int *num_degrees, const int procs, const bool is_cpu, const bool enable_grid_s);
extern __global__ void ODP_Clear_buffers(uint64_t* __restrict__ A, uint64_t* __restrict__ B, const int length);
extern __global__ void ODP_Popcnt(const uint64_t* __restrict__ B, const int nodes,
				  const unsigned int elements, uint64_t* __restrict__ result);
extern __global__ void ODP_Matmul_cuda(const uint64_t* __restrict__ A, uint64_t* __restrict__ B, const int nodes, const int height, const int degree,
				       const int* __restrict__ num_degrees, const int* __restrict__ adjacency, const int* __restrict__ itable,
				       const unsigned int elements, const int symmetries, const int enable_grid_s);
extern __global__ void ODP_Matmul_CHUNK_cuda(const uint64_t* __restrict__ A, uint64_t* __restrict__ B, const int nodes, const int _height, const int degree,
					     const int* __restrict__ num_degrees, const int* __restrict__ adjacency, const int* __restrict__ itable,
					     const int symmetries, const bool enable_grid_s);

static __global__ void init_buffers(uint64_t* __restrict__ A, uint64_t* __restrict__ B, const int nodes,
				    const int symmetries, const unsigned int elements, const int height, const bool enable_grid_s)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < nodes/symmetries) {
    unsigned int offset = tid*elements+tid/UINT64_BITS;
    A[offset] = B[offset] = (0x1ULL<<(tid%UINT64_BITS));
    tid += blockDim.x * gridDim.x;
  }
}

static __global__ void init_buffers_saving(uint64_t* __restrict__ A, uint64_t* __restrict__ B, const int nodes,
					   const int symmetries, const int t, const int height, const bool enable_grid_s)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid<UINT64_BITS*GPU_CHUNK && UINT64_BITS*t*GPU_CHUNK+tid<nodes/symmetries) {
    unsigned int offset = (UINT64_BITS*t*GPU_CHUNK+tid)*GPU_CHUNK+tid/UINT64_BITS;
    A[offset] = B[offset] = (0x1ULL<<(tid%UINT64_BITS));
    tid += blockDim.x * gridDim.x;
  }
}

static void aspl_cuda_mat(const int* __restrict__ adjacency,
			  int *diameter, long *sum, double *ASPL)
{
  unsigned int elements = (_nodes/_symmetries+(UINT64_BITS-1))/UINT64_BITS;
  *sum = (long)_nodes * (_nodes - 1);
  *diameter = 1;
  ODP_Clear_buffers <<< BLOCKS, THREADS >>> (_A_dev, _B_dev, _nodes*elements);
  init_buffers      <<< BLOCKS, THREADS >>> (_A_dev, _B_dev, _nodes, _symmetries, elements, _height, _enable_grid_s);

  for(int kk=0;kk<_nodes;kk++){
    ODP_Matmul_cuda <<< BLOCKS, THREADS >>> (_A_dev, _B_dev, _nodes, _height, _degree, _num_degrees_dev,
					     _adjacency_dev, _itable_dev, elements, _symmetries, _enable_grid_s);
    ODP_Popcnt      <<< BLOCKS, THREADS >>> (_B_dev, _nodes, elements, _result_dev);
    
    hipMemcpy(_result, _result_dev, sizeof(uint64_t)*BLOCKS, hipMemcpyDeviceToHost);
    uint64_t num = 0;
    for (int i=0;i<BLOCKS;i++)
      num += _result[i];

    num *= _symmetries;
    if(num == (uint64_t)_nodes*_nodes) break;

    // swap A <-> B
    uint64_t* tmp = _A_dev;
    _A_dev = _B_dev;
    _B_dev = tmp;

    *sum += (long)_nodes * _nodes - num;
    (*diameter) += 1;
  }
  
  *ASPL = *sum / (((double)_nodes-1)*_nodes);
  *sum /= 2.0;
}

static void aspl_cuda_mat_saving(const int* __restrict__ adjacency,
				 int *diameter, long *sum, double *ASPL)
{
  unsigned int elements = (_nodes/_symmetries+UINT64_BITS-1)/UINT64_BITS;
  int parsize = (elements + GPU_CHUNK - 1)/GPU_CHUNK;
  *sum = (long)_nodes * (_nodes - 1);
  *diameter = 1;

  for(int t=0;t<parsize;t++){
    unsigned int kk, l;
    for(l=0; l<UINT64_BITS*GPU_CHUNK && UINT64_BITS*t*GPU_CHUNK+l<_nodes/_symmetries; l++){}
    ODP_Clear_buffers   <<< BLOCKS, THREADS >>> (_A_dev, _B_dev, _nodes*GPU_CHUNK);
    init_buffers_saving <<< BLOCKS, THREADS >>> (_A_dev, _B_dev, _nodes, _symmetries, t, _height, _enable_grid_s);

    for(kk=0;kk<_nodes;kk++){
      ODP_Matmul_CHUNK_cuda <<< BLOCKS, THREADS >>> (_A_dev, _B_dev, _nodes, _height, _degree, _num_degrees_dev,
						     _adjacency_dev, _itable_dev, _symmetries, _enable_grid_s);
      ODP_Popcnt            <<< BLOCKS, THREADS >>> (_B_dev, _nodes, GPU_CHUNK, _result_dev);

      hipMemcpy(_result, _result_dev, sizeof(uint64_t)*BLOCKS, hipMemcpyDeviceToHost);
      uint64_t num = 0;
      for (int i=0;i<BLOCKS;i++)
        num += _result[i];

      if(num == (uint64_t)_nodes*l) break;

      // swap A <-> B
      uint64_t* tmp = _A_dev;
      _A_dev = _B_dev;
      _B_dev = tmp;
      
      *sum += ((long)_nodes * l - num) * _symmetries;
    }
    *diameter = MAX(*diameter, kk+1);
  }

  *ASPL = *sum / (((double)_nodes-1)*_nodes);
  *sum /= 2.0;
}

static void init_aspl_cuda_s(const int nodes, const int degree,
			     const int* __restrict__ num_degrees, const int symmetries)
{
  hipInit(0);

  if(nodes % symmetries != 0)
    ERROR("nodes(%d) must be divisible by symmetries(%d)\n", nodes, symmetries);

  _kind = ODP_Get_kind(nodes, degree, num_degrees, symmetries, 1, false, _enable_grid_s);
  _mem_usage = ODP_Get_mem_usage(_kind, nodes, degree, symmetries, num_degrees, 1, false, _enable_grid_s);
  size_t s = (_kind == ASPL_MATRIX)? (nodes/symmetries+(UINT64_BITS-1))/UINT64_BITS : GPU_CHUNK;
  s *= nodes * sizeof(uint64_t);

  _nodes = nodes;
  _degree = degree;
  _symmetries = symmetries;
  
  hipMalloc((void**)&_A_dev, s);
  hipMalloc((void**)&_B_dev, s);
  hipHostAlloc((void**)&_result,     sizeof(uint64_t)*BLOCKS, hipHostMallocDefault);
  hipMalloc((void**)&_result_dev,    sizeof(uint64_t)*BLOCKS);
  hipMalloc((void**)&_adjacency_dev, sizeof(int)*(nodes/symmetries)*degree);
  _is_profile = ODP_Check_profile();
  _elapsed_time = 0;
  _times = 0;
  if(num_degrees){
    hipMalloc((void**)&_num_degrees_dev, sizeof(int)*nodes);
    hipMemcpy(_num_degrees_dev, num_degrees, sizeof(int)*nodes, hipMemcpyHostToDevice);
  }
}

extern "C" void ODP_Init_aspl_cuda_general(const int nodes, const int degree, const int* num_degrees)
{
  init_aspl_cuda_s(nodes, degree, num_degrees, 1);
}

extern "C" void ODP_Init_aspl_cuda_general_s(const int nodes, const int degree, const int* num_degrees, const int symmetries)
{
  if(num_degrees){
    int *tmp_num_degrees = (int *)malloc(sizeof(int) * nodes);
    int based_nodes = nodes/symmetries;
    for(int i=0;i<symmetries;i++)
      for(int j=0;j<based_nodes;j++)
        tmp_num_degrees[i*based_nodes+j] = num_degrees[j];
    
    init_aspl_cuda_s(nodes, degree, tmp_num_degrees, symmetries);
    free(tmp_num_degrees);
  }
  else{
    init_aspl_cuda_s(nodes, degree, NULL, symmetries);
  }
}

extern "C" void ODP_Init_aspl_cuda_grid(const int width, const int height, const int degree, const int* num_degrees)
{
  int nodes = width * height;
  _height = height;
  init_aspl_cuda_s(nodes, degree, num_degrees, 1);
}

extern "C" void ODP_Init_aspl_cuda_grid_s(const int width, const int height, const int degree, const int* num_degrees, const int symmetries)
{
  int nodes = width * height;
  _height = height;
  if(symmetries == 2 || symmetries == 4)
    _enable_grid_s = true;
  
  if(num_degrees){
    int *tmp_num_degrees = (int *)malloc(sizeof(int) * nodes);
    int based_nodes = nodes/symmetries;
    if(symmetries == 2){
      for(int i=0;i<based_nodes;i++){
        tmp_num_degrees[i] = num_degrees[i];
        tmp_num_degrees[ODP_ROTATE(i, width, height, symmetries, 180)] = num_degrees[i];
      }
    }
    else if(symmetries == 4){
      for(int i=0;i<based_nodes;i++){
        int v = ODP_LOCAL_INDEX_GRID(i,width,height,symmetries);
        tmp_num_degrees[v] = num_degrees[i];
        tmp_num_degrees[ODP_ROTATE(v, width, height, symmetries,  90)] = num_degrees[i];
        tmp_num_degrees[ODP_ROTATE(v, width, height, symmetries, 180)] = num_degrees[i];
        tmp_num_degrees[ODP_ROTATE(v, width, height, symmetries, 270)] = num_degrees[i];
      }
    }
    init_aspl_cuda_s(nodes, degree, tmp_num_degrees, symmetries);
    free(tmp_num_degrees);
  }
  else{
    init_aspl_cuda_s(nodes, degree, NULL, symmetries);
  }

  if(symmetries > 1){
    hipHostAlloc((void**)&_itable, sizeof(int)*nodes, hipHostMallocDefault);
    ODP_Create_itable(width, height, symmetries, _itable);
    hipMalloc((void**)&_itable_dev, sizeof(int)*nodes);
    hipMemcpy(_itable_dev, _itable, sizeof(int)*nodes, hipMemcpyHostToDevice);
  }
}

extern "C" void ODP_Finalize_aspl()
{
  hipFree(_A_dev);
  hipFree(_B_dev);
  hipHostFree(_result);
  hipFree(_result_dev);
  hipFree(_adjacency_dev);
  if(_num_degrees_dev) hipFree(_num_degrees_dev);
  if(_itable)          hipHostFree(_itable);
  if(_itable_dev)      hipFree(_itable_dev);

  if(_is_profile)
    ODP_Profile("CUDA", _kind, _symmetries, _mem_usage,
		_elapsed_time, _times, 1);
}

extern "C" void ODP_Set_aspl(const int* __restrict__ adjacency,
			     int *diameter, long *sum, double *ASPL)
{
  double t = ODP_Get_time();
  
  hipMemcpy(_adjacency_dev, adjacency, sizeof(int)*(_nodes/_symmetries)*_degree, hipMemcpyHostToDevice);
  
  if(_kind == ASPL_MATRIX)
    aspl_cuda_mat       (adjacency, diameter, sum, ASPL);
  else
    aspl_cuda_mat_saving(adjacency, diameter, sum, ASPL);

  _elapsed_time += ODP_Get_time() - t;
    
  if(*diameter > _nodes){
    *diameter = INT_MAX;
    *sum = LONG_MAX;
    *ASPL = DBL_MAX;
  }

  _times++;
}
