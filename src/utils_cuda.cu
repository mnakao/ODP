#include "hip/hip_runtime.h"
#include "common.h"

__global__ void create_adjacency(const int based_elements, const int total_elements,
				 const int based_nodes, const int nodes, int* __restrict__ adjacency_dev)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x + based_elements;
  while (tid < total_elements) {
    int t = tid/based_elements;
    int i = tid - (t*based_elements);
    int v = adjacency_dev[i] + t*based_nodes;
    adjacency_dev[tid] = (v < nodes)? v : v - nodes;

    tid += blockDim.x * gridDim.x;
  }
}

__global__ void clear_buffers(uint64_t* __restrict__ A, uint64_t* __restrict__ B, const int length)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid<length) {
    A[tid] = B[tid] = 0;
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void popcnt(const uint64_t* __restrict__ B, const int nodes,
		       const unsigned int elements, uint64_t* __restrict__ result)
{
  __shared__ uint64_t cache[THREADS];
  int cacheIndex = threadIdx.x;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  uint64_t num = 0;
  while (tid < elements*nodes) {
    num += POPCNT(B[tid]);
    tid += blockDim.x * gridDim.x;
  }
  cache[cacheIndex] = num;
  __syncthreads();

  int i = blockDim.x/2;
  while (i != 0){
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex+i];
    __syncthreads();
    i /= 2;
  }

  if(cacheIndex == 0)
    result[blockIdx.x] = cache[0];
}

__global__ void matrix_op(const uint64_t* __restrict__ A, uint64_t* __restrict__ B, const int* __restrict__ adjacency,
			  const int* __restrict__ num_degrees, const int nodes, const int degree, const unsigned int elements)
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if(!num_degrees){
    while (tid < nodes*elements) {
      int i = tid / elements;
      int k = tid % elements;
      uint64_t tmp = B[tid];
      for(int j=0;j<degree;j++){
        int n = *(adjacency + i * degree + j);  // int n = adjacency[i][j];
        tmp |= A[n*elements+k];
      }
      B[tid] = tmp;
      tid += blockDim.x * gridDim.x;
    }
  }
  else{
    while (tid < nodes*elements) {
      int i = tid / elements;
      int k = tid % elements;
      uint64_t tmp = B[tid];
      for(int j=0;j<num_degrees[i];j++){
        int n = *(adjacency + i * degree + j);  // int n = adjacency[i][j];
        tmp |= A[n*elements+k];
      }
      B[tid] = tmp;
      tid += blockDim.x * gridDim.x;
    }
  }
}

